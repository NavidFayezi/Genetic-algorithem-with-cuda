#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>



#define thrdperblc 128
#define POPULATION 10000



__global__ void crossover(char* population) {
    int i, first_start, second_start, first_end, second_end, iterator,endofpopulation;
    int tId = blockIdx.x * thrdperblc + threadIdx.x;
    if (tId <= POPULATION / 2) {
        iterator = 0;
        first_start = tId * 11 * 2;
        first_end = first_start + 7;
        endofpopulation =  (tId / 2 * 11) + POPULATION * 11;
        for (i = first_start; i <first_end ;i++) {
            population[ endofpopulation + iterator] = population[i];
            iterator++;
        }
        second_start = first_start + 18;
        second_end = second_start + 4;
        for (i = second_start; i < second_end ; i++) {
            population[endofpopulation + iterator] = population[i];
            iterator++;
        }
    }
}

__global__ void mutatation(char* population) {
    int i,start,end;
    int tId = blockIdx.x * thrdperblc + threadIdx.x;
    if (tId <= POPULATION / 2) {
        hiprandState state;
        char letters[] = " ABCDEFGHIJKLMNOPQRSTUVWXYZ";
        start = POPULATION * 11 + (tId * 11);
        end = start + 11;
        for (i = start; i < end ; i++) {
            hiprand_init((unsigned long long)clock() + tId + 1, 0, 0, &state);
            int randnum = (int)( 834 * hiprand_uniform_double(&state));
            if ((int)randnum % 10 > 7) {
                population[i] = letters[randnum % 27];
            }
        }
    }

}

int fitness(char* chr) {
    int i = 0;
    int fitness_val = 0;
    char word[] = "HELLO WORLD";
    for (i = 0; i < 11; i++) {
        if(chr[i] == word[i])
            fitness_val += 5;
        else {
            fitness_val -= abs(chr[i] - word[i]);
        }
    }
    return fitness_val;
}

int compare(const void* first, const void* second)
{
    char* f = ((char*)first);
    char* s = ((char*)second);
    int temp = fitness(s) - fitness(f);
    return temp;
}

void init_population(char* population) {
    char letters[] = " ABCDEFGHIJKLMNOPQRSTUVWXYZ";
    int i;
    for (i = 0; i < POPULATION * 11; i++) {
        population[i] = letters[rand() % 27];
    }

}

int main()
{
    int i;
    float time;
    hipEvent_t start, stop;
    char* population = (char*)malloc(sizeof(char) * (POPULATION + POPULATION / 2) * 11 );
    char* d_population;
    hipMalloc(&d_population, sizeof(char) * (POPULATION + POPULATION / 2) * 11 );
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    init_population(population);
    qsort(population, POPULATION, sizeof(char) * 11, compare);
    while (fitness(population) != 55) {
        hipMemcpy(d_population, population, sizeof(char) * 11 * (POPULATION), hipMemcpyHostToDevice);
        crossover << < POPULATION/ 2 / thrdperblc  + 1, thrdperblc >> > (d_population);
        mutatation << < POPULATION / 2 / thrdperblc + 1, thrdperblc >> > (d_population);
        hipMemcpy(population, d_population, sizeof(char) * 11 * (POPULATION + POPULATION/2), hipMemcpyDeviceToHost);
        qsort(population, POPULATION + POPULATION / 2, sizeof(char) * 11, compare);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    for (i = 0; i < 11; i++)
        printf("%c", population[i]);
    printf("\n");
    printf("execution time is : %f miliseconds \n", time);
    hipFree(d_population);
    free(population);
    return 0;
}
